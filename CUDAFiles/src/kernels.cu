#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "stdio.h"

//This include is completely unnecessary and can be omitted - only used to prevent Intellisense from thinking CUDA variables are undefined
#include <>

__global__ void propagator(int N, int M, double z, double dx, double n, double lambda, hipfftComplex* Hq){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float FX, FY, res;
    float pre = (float)(n/lambda);
    float calc = (float)(1/dx);
    int newIndex;
    int count = N*M;
    for (int i = index; i < count; i += stride)
    {
        newIndex = (i + count/2-1) % (count);
        FX = ((float)(1+(i/M)) * calc/(float)(N)) - calc/2.0f;
        FY = ((float)(1+(i%M)) * calc/(float)(M)) - calc/2.0f;
        res = 2 * (float)(M_PI*z*pre) * sqrtf(1 - SQUARE(FX/pre) - SQUARE(FY/pre));
        if (sqrtf(SQUARE(FX)+SQUARE(FX)) < pre){
            Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipFloatComplex(cosf(res),sinf(res));
        }else{
            Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipFloatComplex(0,0);
        }
    }
}

__global__ void multiply(int count, hipfftComplex*  in, hipfftComplex* out){
    hipfftComplex temp;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        temp = make_hipFloatComplex(out[i].x/(float)(count), out[i].y/(float)(count));
        out[i] = hipCmulf(in[i], temp);
    }
}

__global__ void multiplyf(int count, double*  in1, double*  in2, double*  out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = in1[i]*in2[i];
    }
}

__global__ void multiplyfc(int count, double* in, hipfftDoubleComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = make_hipDoubleComplex(out[i].x*in[i],out[i].y*in[i]);
    }
}

__global__ void absolute(int count, hipfftDoubleComplex* in, double* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCabs(in[i]);
    }
}

__global__ void real(int count, hipfftDoubleComplex* in, double* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = in[i].x;
    }
}

__global__ void imag(int count, hipfftDoubleComplex* in, double* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = in[i].y;
    }
}

__global__ void angle(int count, hipfftDoubleComplex* in, double* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = atan2(in[i].y,in[i].x);
    }
}

//Fast parallel sum
/* 
*   The following function of sum is taken from the publicly accessible NVidia 
*   webinar found at https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
*/
__global__ void sum(int count, double* in, double* result){
    extern __shared__ double sharedIn[];
    int thIdx = threadIdx.x;
    int index = blockIdx.x*blockDim.x + thIdx;
    int stride = blockDim.x*gridDim.x;
    sharedIn[thIdx] = 0;
    
    for(unsigned int i = index; i < count; i+=stride){
        sharedIn[thIdx] += in[i];
    }
    __syncthreads();
    for(unsigned int i = blockDim.x/2 ; i>0 ; i>>=1){
        if(thIdx < i){
            sharedIn[thIdx] += sharedIn[thIdx+i];
        }
        __syncthreads();
    }
    if(thIdx == 0) result[blockIdx.x] = sharedIn[0];
}

__global__ void sumOfProducts(int count, double* in1, double* in2, double* result){
    extern __shared__ double sharedIn[];
    int thIdx = threadIdx.x;
    int index = blockIdx.x*blockDim.x + thIdx;
    int stride = blockDim.x*gridDim.x;
    sharedIn[thIdx] = 0;
    
    for(unsigned int i = index; i < count; i+=stride){
        sharedIn[thIdx] += in1[i]*in2[i];
    }
    __syncthreads();
    for(unsigned int i = blockDim.x/2 ; i>0 ; i>>=1){
        if(thIdx < i){
            sharedIn[thIdx] += sharedIn[thIdx+i];
        }
        __syncthreads();
    }
    if(thIdx == 0) result[blockIdx.x] = sharedIn[0];
}

__global__ void maximum(int count, double* in, double* result){
    result[0] = 0.0f;
    extern __shared__ double sharedIn[];
    int thIdx = threadIdx.x;
    int index = blockIdx.x*blockDim.x + thIdx;
    int stride = blockDim.x*gridDim.x;
    sharedIn[thIdx] = in[index];
    
    for(int i = index+stride; i < count; i += stride){
        sharedIn[thIdx] = fmax(sharedIn[thIdx], in[index]);
    }
    __syncthreads();
    for(unsigned int i = blockDim.x/2 ; i>0 ; i>>=1){
        if(thIdx < i){
            sharedIn[thIdx] =  fmax(sharedIn[thIdx], sharedIn[thIdx+i]);
        }
        __syncthreads();
    }
    if (thIdx == 0) result[0] = sharedIn[thIdx];
}


__global__ void F2C(int count, double*  in, hipfftDoubleComplex*  out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = make_hipDoubleComplex(in[i], 0);
    }
}

__global__ void modelFunc(int count, int numLayers, double rOffset, double iOffset, hipfftDoubleComplex* in, hipfftDoubleComplex* model, double* Imodel){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        model[i] = make_hipDoubleComplex(rOffset, iOffset);
        for(int j = 0; j < numLayers; j++){
            model[i] = hipCadd(model[i], in[i + j*count]);
        }
        Imodel[i] = SQUARE(hipCabs(model[i]));
    }
}

__global__ void conjugate(int count, hipfftComplex *in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipConjf(in[i]);
    }
}

__global__ void simpleDivision(double* num, double* div, double* res){
        if(div[0] == 0.0f)
            div[0] = div[0] + 0.00001f;
        res[0] = num[0] / div[0];
}

__global__ void linear(int count, double* coef, double* constant, double* in, double* out, bool sign){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        if(sign)
            out[i] = fma(coef[0], in[i], constant[i]);
        else
            out[i] = fma(coef[0], in[i], -constant[i]);
    }
}

__global__ void square(int count, double* in, double* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = SQUARE(in[i]);
    }
}

__global__ void simpleSum(double* in1, double* in2, double* out){
    out[0] = in1[0] + in2[0];
}

__global__ void cMultiplyf(int count, double constant, double* in, double* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = constant*in[i];
    }
}

__global__ void cMultiply(int count, hipfftDoubleComplex* constant, hipfftDoubleComplex* in, hipfftDoubleComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCmul(constant[0],in[i]);
    }
}

__global__ void cMultiplyfc(int count, double constant, hipfftDoubleComplex* in, hipfftDoubleComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = make_hipDoubleComplex(in[i].x*constant, in[i].y*constant);
    }
}

__global__ void cMultiplyfcp(int count, double *constant, hipfftDoubleComplex* in, hipfftDoubleComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = make_hipDoubleComplex(in[i].x*constant[0], in[i].y*constant[0]);
    }
}

__global__ void cDividefp(int count, double *constant, double* in, double* out){
    if(constant[0] == 0.0f){
        constant[0] += 0.00001;
    }
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = in[i] / constant[0];
    }
}

__global__ void add(int count, hipfftDoubleComplex* in1, hipfftDoubleComplex* in2, hipfftDoubleComplex* out, bool sign){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        if (sign)
            out[i] = hipCadd(in1[i], in2[i]);
        else
            out[i] = hipCsub(in1[i], in2[i]); 
    }
}

__global__ void strictBounds(int count, hipfftDoubleComplex* arr, double r_min, double r_max, double i_min, double i_max){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        arr[i].x = fmax(fmin(r_max, arr[i].x), r_min);
        arr[i].y = fmax(fmin(i_max, arr[i].y), i_min);
    }
}

__global__ void softBounds(int count, hipfftDoubleComplex* arr, double mu, double t){
    double tmp = mu*t;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        hipfftDoubleComplex temp = make_hipDoubleComplex(arr[i].x-tmp,arr[i].y);
        if(hipCabs(temp) < 0)
            arr[i] = make_hipDoubleComplex(0,0);
        else
            arr[i] = temp;
    }
}

// Most naive implementation of gaussian bluring - only effective on very small kernel sizes
// Future implementation could use shared memory for larger bandwidth
__global__ void rowConvolution(int N, int M, double diameter, double* kernel, double* image, double* output, bool horizontal){
    int offset;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;
    int count = N*M;
    for(int i = index; i < count; i+=gridSize){
        output[i] = 0;
        for(int j = 0; j < diameter; j++){
            offset = j - diameter/2;
            if(horizontal){
                if((i%N)+offset >= 0 && (i%N)+offset < N){
                    output[i] += kernel[j]*image[i+offset];
                }
                else
                    output[i] += kernel[j];
            } else {
                if((i/M)+offset >= 0 && (i/M)+offset < M){
                    output[i] += kernel[j]*image[i+offset*M];
                }
                else
                    output[i] += kernel[j];
            }
        }
    }



}

__global__ void offset(int count, double roff, double ioff, hipfftDoubleComplex* in, hipfftDoubleComplex* out){
    hipfftDoubleComplex temp = make_hipDoubleComplex(roff, ioff);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCadd(temp, in[i]);
    }
}

__global__ void offsetf(int count, double roff, double* in, double* out, bool sign){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        if(sign)
            out[i] = roff + in[i];
        else
            out[i] = roff - in[i];
    }
}

__global__ void C2Z(int count, hipfftComplex* in, hipfftDoubleComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = make_hipDoubleComplex((double)in[i].x, (double)in[i].y);
    }
}

__global__ void Z2C(int count, hipfftDoubleComplex* in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = make_hipFloatComplex((float)in[i].x, (float)in[i].y);
    }
}

__global__ void extend(int count, int multiple, hipfftDoubleComplex* in, hipfftDoubleComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        for(int e = 0; e < multiple; e++){
            out[i + e*count] = in[i];
        }
    }
}
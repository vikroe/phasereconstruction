#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

__global__ void propagator(int N, int M, float z, float dx, float n, float lambda, hipfftComplex* Hq){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float FX, FY, temp, res;
    float pre = n/lambda;
    float calc = 1/dx;
    int newIndex;
    int count = N*M;
    for (int i = index; i < count; i += stride)
    {
        newIndex = (i + count/2-1) % (count);
        FX = ((float)(1+(i/M)) * calc/(float)(N)) - calc/2.0f;
        FY = ((float)(1+(i%M)) * calc/(float)(M)) - calc/2.0f;
        res = 2 * M_PI*z*pre * sqrt(1 - SQUARE(FX/pre) - SQUARE(FY/pre));
        if(temp == 0.0){
            Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(0,0);
        }
        else{
            Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(std::cos(res),std::sin(res));
        }
    }
}

__global__ void multiply(int N, int M, hipfftComplex*  in, hipfftComplex* out){
    hipfftComplex temp;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        temp = make_hipFloatComplex(in[i].x/(float)(N*M), in[i].y/(float)(N*M));
        out[i] = hipCmulf(out[i], temp);
    }
}

__global__ void multiplyf(int N, int M, float*  in1, float*  in2, float*  out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = in1[i]*in2[i];
    }
}

__global__ void multiplyfc(int count, float* in, hipfftComplex* out){
    hipfftComplex temp;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        temp = make_hipFloatComplex(in[i], 0);
        out[i] = hipCmulf(temp,out[i]);
    }
}

__global__ void absolute(int N, int M, hipfftComplex* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = hipCabsf(in[i]);
    }
}

__global__ void angle(int count, hipfftComplex* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = atan2f(in[i].y,in[i].x);
    }
}


__device__ void warpReduce(volatile float *sdata, int thIdx){
    if(N_THREADS>=64) sdata[thIdx] += sdata[thIdx + 32];
    if(N_THREADS>=32) sdata[thIdx] += sdata[thIdx + 16];
    if(N_THREADS>=16) sdata[thIdx] += sdata[thIdx + 8];
    if(N_THREADS>=8) sdata[thIdx] += sdata[thIdx + 4];
    if(N_THREADS>=4) sdata[thIdx] += sdata[thIdx + 2];
    if(N_THREADS>=2) sdata[thIdx] += sdata[thIdx + 1];
}

//Fast parallel sum 
__global__ void sum(int count, float* in, float* result){
    extern __shared__ float sharedIn[];
    int thIdx = threadIdx.x;
    int index = blockIdx.x*(N_THREADS*2) + thIdx;
    int stride = N_THREADS*2*gridDim.x;
    sharedIn[thIdx] = 0;
    
    while(index < count){
        sharedIn[thIdx] += in[index] + in[index+N_THREADS];
        index += stride;
    }
    __syncthreads();

    if (N_THREADS >= 512){
        if (thIdx < 256){
            sharedIn[thIdx] += sharedIn[thIdx + 256]; 
        } 
        __syncthreads();
    }
    if (N_THREADS >= 256){
        if (thIdx < 128){
            sharedIn[thIdx] += sharedIn[thIdx + 128];
        }
        __syncthreads();
    }
    if (N_THREADS >= 128){
        if (thIdx <  64){
            sharedIn[thIdx] += sharedIn[thIdx +  64];
        }
        __syncthreads();
    }
    if (thIdx < 32) warpReduce(sharedIn, thIdx);
    if (thIdx == 0) result[blockIdx.x] = sharedIn[0];
}

__global__ void F2C(int N, int M, float*  in, hipfftComplex*  out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = make_hipFloatComplex(in[i], 0);
    }
}

__global__ void modelFunc(int N, int M, int count, float rOffset, float iOffset, hipfftComplex* in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = make_hipFloatComplex(rOffset, iOffset);
        for(int j = 0; j < count; j++){
            out[i] = hipCaddf(out[i], in[i + i*N*M]);
        }
    }
}

__global__ void ImodelFunc(int N, int M, hipfftComplex* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = SQUARE(hipCabsf(in[i]));
    }
}

__global__ void conjugate(int N, int M, hipfftComplex *in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = hipConjf(in[i]);
    }
}

__global__ void simpleDivision(float* num, float* div, float* res){
        int i = threadIdx.x;
        res[i] = num[i] / div[i];
}

__global__ void linear(int N, int M, float* coef, float* constant, float* in, float* out, bool sign){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        if(sign)
            out[i] = coef[0]*in[i] + constant[i];
        else
            out[i] = coef[0]*in[i] - constant[i];
    }
}

__global__ void square(int count, float* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = SQUARE(in[i]);
    }
}

__global__ void simpleSum(float* in1, float* in2, float* out){
    int i = threadIdx.x;
    out[i] = in1[i] + in2[i];
}

__global__ void cMultiplyf(int count, float constant, float* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = constant*in[i];
    }
}

__global__ void cMultiply(int count, hipfftComplex* constant, hipfftComplex* in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCmulf(constant[0],in[i]);
    }
}

__global__ void cMultiplyfc(int count, float constant, hipfftComplex* in, hipfftComplex* out){
    hipfftComplex temp = make_hipFloatComplex(constant, 0);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCmulf(temp,in[i]);
    }
}

__global__ void cMultiplyfcp(int count, float *constant, hipfftComplex* in, hipfftComplex* out){
    hipfftComplex temp = make_hipFloatComplex(constant[0], 0);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCmulf(temp,in[i]);
    }
}

__global__ void add(int count, hipfftComplex* in1, hipfftComplex* in2, hipfftComplex* out, bool sign){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        if (sign)
            out[i] = hipCaddf(in1[i], in2[i]);
        else
            out[i] = hipCsubf(in1[i], in2[i]); 
    }
}

__global__ void strictBounds(int count, hipfftComplex* arr, float r_min, float r_max, float i_min, float i_max){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        if (arr[i].x < r_min)
            arr[i].x = r_min;
        else if (arr[i].x > r_max)
            arr[i].x = r_max;
        if (arr[i].y < i_min)
            arr[i].y = i_min;
        else if (arr[i].y > i_max)
            arr[i].y = i_max; 
    }
}

__global__ void softBounds(int count, hipfftComplex* arr, float mu, float t){
    hipfftComplex zero = make_hipFloatComplex(0,0);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        hipfftComplex temp = make_hipFloatComplex(arr[i].x-mu*t,arr[i].y);
        if(hipCabsf(temp) < 0)
            arr[i] = zero;
        else
            arr[i] = temp;
    }
}

#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

__global__ void propagator(int N, int M, float z, float dx, float n, float lambda, hipfftComplex* Hq){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float FX, FY, temp, res;
    float pre = n/lambda;
    float calc = 1/dx;
    int newIndex;
    int count = N*M;
    for (int i = index; i < count; i += stride)
    {
        newIndex = (i + count/2-1) % (count);
        FX = ((float)(1+(i/M)) * calc/(float)(N)) - calc/2.0f;
        FY = ((float)(1+(i%M)) * calc/(float)(M)) - calc/2.0f;
        res = 2 * M_PI*z*pre * sqrt(1 - SQUARE(FX/pre) - SQUARE(FY/pre));
        if(temp == 0.0){
            Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(0,0);
        }
        else{
            Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(std::cos(res),std::sin(res));
        }
    }
}

__global__ void multiply(int N, int M, hipfftComplex*  in, hipfftComplex*  out){
    hipfftComplex temp;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        temp = make_hipFloatComplex(out[i].x/(float)(N*M), out[i].y/(float)(N*M));
        out[i] = hipCmulf(in[i], temp);
    }
}

__global__ void F2C(int N, int M, float*  in, hipfftComplex*  out){
    hipfftComplex temp;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = make_hipFloatComplex(in[i], 0);
    }
}

__global__ void modelFunc(int N, int M, int count, float rOffset, float iOffset, hipfftComplex* in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = make_hipFloatComplex(rOffset, iOffset);
        for(int j = 0; j < count; j++){
            out[i] = hipCaddf(out[i], in[i + i*N*M]);
        }
    }
}

__global__ void ImodelFunc(int N, int M, hipfftComplex* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = SQUARE(hipCabsf(in[i]));
    }
}

__global__ void conjugate(int N, int M, hipfftComplex *in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N*M; i += stride){
        out[i] = hipConjf(in[i]);
    }
}
#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

//This include is completely unnecessary and can be omitted - only used to prevent Intellisense from thinking CUDA variables are undefined
#include <>

__global__ void propagator(int N, int M, float z, float dx, float n, float lambda, hipfftComplex* Hq){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    float FX, FY, temp, res;
    float pre = n/lambda;
    float calc = 1/dx;
    int newIndex;
    int count = N*M;
    for (int i = index; i < count; i += stride)
    {
        newIndex = (i + count/2-1) % (count);
        FX = ((float)(1+(i/M)) * calc/(float)(N)) - calc/2.0f;
        FY = ((float)(1+(i%M)) * calc/(float)(M)) - calc/2.0f;
        res = 2 * M_PI*z*pre * sqrtf(1 - SQUARE(FX/pre) - SQUARE(FY/pre));
        Hq[(newIndex % M) > M/2-1 ? newIndex-M/2 : newIndex+M/2] = make_hipComplex(cosf(res),sinf(res));
    }
}

__global__ void multiply(int count, hipfftComplex*  in, hipfftComplex* out){
    hipfftComplex temp;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        temp = make_hipFloatComplex(in[i].x/(float)(count), in[i].y/(float)(count));
        out[i] = hipCmulf(out[i], temp);
    }
}

__global__ void multiplyf(int count, float*  in1, float*  in2, float*  out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = in1[i]*in2[i];
    }
}

__global__ void multiplyfc(int count, float* in, hipfftComplex* out){
    hipfftComplex temp;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        temp = make_hipFloatComplex(in[i], 0);
        out[i] = hipCmulf(temp,out[i]);
    }
}

__global__ void absolute(int count, hipfftComplex* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCabsf(in[i]);
    }
}

__global__ void angle(int count, hipfftComplex* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = atan2f(in[i].y,in[i].x);
    }
}


__device__ void warpReduce(volatile float *sdata, int thIdx){
    if(N_THREADS>=64) sdata[thIdx] += sdata[thIdx + 32];
    if(N_THREADS>=32) sdata[thIdx] += sdata[thIdx + 16];
    if(N_THREADS>=16) sdata[thIdx] += sdata[thIdx + 8];
    if(N_THREADS>=8) sdata[thIdx] += sdata[thIdx + 4];
    if(N_THREADS>=4) sdata[thIdx] += sdata[thIdx + 2];
    if(N_THREADS>=2) sdata[thIdx] += sdata[thIdx + 1];
}

//Fast parallel sum 
__global__ void sum(int count, float* in, float* result){
    extern __shared__ float sharedIn[];
    int thIdx = threadIdx.x;
    int index = blockIdx.x*(N_THREADS*2) + thIdx;
    int stride = N_THREADS*2*gridDim.x;
    sharedIn[thIdx] = 0;
    
    while(index < count){
        sharedIn[thIdx] += in[index] + in[index+N_THREADS];
        index += stride;
    }
    __syncthreads();

    if (N_THREADS >= 512){
        if (thIdx < 256){
            sharedIn[thIdx] += sharedIn[thIdx + 256]; 
        } 
        __syncthreads();
    }
    if (N_THREADS >= 256){
        if (thIdx < 128){
            sharedIn[thIdx] += sharedIn[thIdx + 128];
        }
        __syncthreads();
    }
    if (N_THREADS >= 128){
        if (thIdx <  64){
            sharedIn[thIdx] += sharedIn[thIdx +  64];
        }
        __syncthreads();
    }
    if (thIdx < 32) warpReduce(sharedIn, thIdx);
    if (thIdx == 0) result[blockIdx.x] = sharedIn[0];
}

__global__ void maximum(int count, float* in, float* result){
    extern __shared__ float sharedIn[];
    int thIdx = threadIdx.x;
    int index = blockIdx.x*(N_THREADS*2) + thIdx;
    int stride = N_THREADS*2*gridDim.x;
    sharedIn[thIdx] = 0;
    
    while(index < count){
        sharedIn[thIdx] = (in[index] > in[index+N_THREADS]) ? in[index] : in[index+N_THREADS];
        index += stride;
    }
    __syncthreads();

    for(unsigned int i = blockDim.x/2 ; i>0 ; i>>=1){
        if(thIdx < i){
            sharedIn[thIdx] = (sharedIn[thIdx] > sharedIn[thIdx+i]) ? sharedIn[thIdx] : sharedIn[thIdx+i];
        }
        __syncthreads();
    }
    if (thIdx == 0) result[blockIdx.x] = sharedIn[0];
}


__global__ void F2C(int count, float*  in, hipfftComplex*  out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = make_hipFloatComplex(in[i], 0);
    }
}

__global__ void modelFunc(int count, int numLayers, float rOffset, float iOffset, hipfftComplex* in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = make_hipFloatComplex(rOffset, iOffset);
        for(int j = 0; j < numLayers; j++){
            out[i] = hipCaddf(out[i], in[i + j*count]);
        }
    }
}

__global__ void ImodelFunc(int count, hipfftComplex* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = SQUARE(hipCabsf(in[i]));
    }
}

__global__ void conjugate(int count, hipfftComplex *in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipConjf(in[i]);
    }
}

__global__ void simpleDivision(float* num, float* div, float* res){
        if(div[0] == 0.0f)
            div[0] = div[0] + 0.00001f;
        res[0] = num[0] / div[0];
}

__global__ void linear(int count, float* coef, float* constant, float* in, float* out, bool sign){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        if(sign)
            out[i] = coef[0]*in[i] + constant[i];
        else
            out[i] = coef[0]*in[i] - constant[i];
    }
}

__global__ void square(int count, float* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = SQUARE(in[i]);
    }
}

__global__ void simpleSum(float* in1, float* in2, float* out){
    out[0] = in1[0] + in2[0];
}

__global__ void cMultiplyf(int count, float constant, float* in, float* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = constant*in[i];
    }
}

__global__ void cMultiply(int count, hipfftComplex* constant, hipfftComplex* in, hipfftComplex* out){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCmulf(constant[0],in[i]);
    }
}

__global__ void cMultiplyfc(int count, float constant, hipfftComplex* in, hipfftComplex* out){
    hipfftComplex temp = make_hipFloatComplex(constant, 0);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCmulf(temp,in[i]);
    }
}

__global__ void cMultiplyfcp(int count, float *constant, hipfftComplex* in, hipfftComplex* out){
    hipfftComplex temp = make_hipFloatComplex(constant[0], 0);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = hipCmulf(temp,in[i]);
    }
}

__global__ void cDividefp(int count, float *constant, float* in, float* out){
    if(constant[0] == 0.0f){
        constant[0] += 0.00001;
    }
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        out[i] = in[i] / constant[0];
    }
}

__global__ void add(int count, hipfftComplex* in1, hipfftComplex* in2, hipfftComplex* out, bool sign){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        if (sign)
            out[i] = hipCaddf(in1[i], in2[i]);
        else
            out[i] = hipCsubf(in1[i], in2[i]); 
    }
}

__global__ void strictBounds(int count, hipfftComplex* arr, float r_min, float r_max, float i_min, float i_max){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        if (arr[i].x < r_min)
            arr[i].x = r_min;
        else if (arr[i].x > r_max)
            arr[i].x = r_max;
        if (arr[i].y < i_min)
            arr[i].y = i_min;
        else if (arr[i].y > i_max)
            arr[i].y = i_max; 
    }
}

__global__ void softBounds(int count, hipfftComplex* arr, float mu, float t){
    hipfftComplex zero = make_hipFloatComplex(0,0);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < count; i += stride){
        hipfftComplex temp = make_hipFloatComplex(arr[i].x-mu*t,arr[i].y);
        if(hipCabsf(temp) < 0)
            arr[i] = zero;
        else
            arr[i] = temp;
    }
}

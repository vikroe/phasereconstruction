#include "multilayer.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "kernels.h"
#include <vector>
#include <iostream>

MultiLayer::MultiLayer(int width, int height, std::vector<float> z, float dx, float lambda, float n) :width{width}, height{height}
{
    numLayers = (int)z.size();
    numBlocks = (width*height/2 + N_BLOCKS -1)/N_BLOCKS;

    hipMalloc(&Hq, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&Hn, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&res, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&guess, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&newGuess, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&u, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&temporary, numLayers*width*height*sizeof(hipfftComplex));

    hipMalloc(&image, width*height*sizeof(hipfftComplex));
    hipMalloc(&imagef, width*height*sizeof(float));
    
    hipMalloc(&model, width*height*sizeof(hipfftComplex));
    hipMalloc(&Imodel, width*height*sizeof(float));

    hipfftPlan2d(&fftPlan, width, height, HIPFFT_C2C);

    multilayerPropagator(z, dx, lambda, n);
}

void MultiLayer::multilayerPropagator(std::vector<float> z, float dx, float lambda, float n){
    hipfftComplex *placeHolder;
    for(int i = 0; i < numLayers; i++){
        placeHolder = &Hq[i*width*height];
        propagator<<<N_BLOCKS,numBlocks>>>(width, height, z[i], dx, n, lambda, placeHolder);
    }
}

void MultiLayer::propagate(hipfftComplex* kernel, hipfftComplex* input, hipfftComplex* out){
    hipfftExecC2C(fftPlan, input, out, HIPFFT_FORWARD);
    multiply<<<N_BLOCKS, numBlocks>>>(width,height,kernel,out);
    hipfftExecC2C(fftPlan, out, out, HIPFFT_BACKWARD);
}

void MultiLayer::iterate(float *input, int iters){
    // Initialization of variables
    s = 1;
    hipfftComplex *placeHolder;
    hipfftComplex *HplaceHolder;

    conjugate<<<N_BLOCKS,numBlocks>>>(width*numLayers, height, Hq, Hn);

    //Allocating the device memory array for cost at each iteration
    hipMalloc(&cost, (1+iters)*sizeof(float));

    //Copying the input image from host to device memory - computationally complex
    hipMemcpy(imagef, input, width*height*sizeof(float), hipMemcpyHostToDevice);
    F2C<<<N_BLOCKS, numBlocks>>>(width, height, imagef, image);

    //Copying the device memory image to device memory guesses
    for(int i = 0; i < numLayers; i++){
        hipMemcpy(&guess[i*width*height], image, hipMemcpyDeviceToDevice);
        hipMemcpy(&u[i*width*height], image, hipMemcpyDeviceToDevice);
    }

    for(int iter = 0; iter < iters; iter++){

        //Calculating the current iteration model
        for(int i = 0; i < numLayers; i++){
            placeHolder = &temporary[i*width*height];
            HplaceHolder = &Hq[i*width*height];
            propagate(HplaceHolder, image, placeHolder);
        }
        modelFunc<<<N_BLOCKS,numBlocks>>>(width, height, numLayers, 1.0f, 0, temporary, model);
        ImodelFunc<<<N_BLOCKS,numBlocks>>>(width,height,model,Imodel);

        
    
    }

    // Deallocation of variables
    hipFree(cost);
}

MultiLayer::~MultiLayer(){
    hipFree(Hq);
    hipFree(Hn);
    hipFree(temporary);
    hipFree(image);
    hipFree(imagef);
    hipFree(res);
    hipFree(model);
    hipFree(guess);
    hipFree(newGuess);
    hipFree(u);
    hipFree(Imodel);
    hipfftDestroy(fftPlan);
}
#include "hip/hip_runtime.h"
#include "multilayer.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "kernels.h"
#include <vector>
#include <iostream>

MultiLayer::MultiLayer(int width, int height, std::vector<float> z, float dx, float lambda, float n) :width{width}, height{height}
{
    numLayers = (int)z.size();
    numBlocks = (width*height/2 + N_BLOCKS -1)/N_BLOCKS;

    hipMalloc(&Hq, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&Hn, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&res, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&guess, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&newGuess, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&u, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&temporary, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&temporaryf, 2*numLayers*width*height*sizeof(float));

    hipMalloc(&image, width*height*sizeof(hipfftComplex));
    hipMalloc(&imagef, width*height*sizeof(float));
    hipMalloc(&sumArr, 2*N_BLOCKS*sizeof(float));
    hipMalloc(&c, sizeof(hipfftComplex));
    
    hipMalloc(&model, width*height*sizeof(hipfftComplex));
    hipMalloc(&Imodel, width*height*sizeof(float));

    hipfftPlan2d(&fftPlan, width, height, HIPFFT_C2C);

    multilayerPropagator(z, dx, lambda, n);
}

void MultiLayer::multilayerPropagator(std::vector<float> z, float dx, float lambda, float n){
    hipfftComplex *placeHolder;
    for(int i = 0; i < numLayers; i++){
        placeHolder = &Hq[i*width*height];
        propagator<<<N_BLOCKS,N_THREADS>>>(width, height, z[i], dx, n, lambda, placeHolder);
    }
}

void MultiLayer::propagate(hipfftComplex* kernel, hipfftComplex* input, hipfftComplex* out){
    hipfftExecC2C(fftPlan, input, out, HIPFFT_FORWARD);
    multiply<<<N_BLOCKS, N_THREADS>>>(width,height,kernel,out);
    hipfftExecC2C(fftPlan, out, out, HIPFFT_BACKWARD);
}

void MultiLayer::iterate(float *input, int iters, float mu, float* rconstr, float* iconstr){
    // Initialization of variables
    s = 1;
    int count = width*height;
    hipfftComplex *placeHolder;
    hipfftComplex *HplaceHolder;

    conjugate<<<N_BLOCKS,N_THREADS>>>(width*numLayers, height, Hq, Hn);

    //Allocating the device memory array for cost at each iteration
    hipMalloc(&cost, (1+iters)*sizeof(float));

    //Copying the input image from host to device memory - computationally complex
    hipMemcpy(imagef, input, width*height*sizeof(float), hipMemcpyHostToDevice);
    F2C<<<N_BLOCKS, N_THREADS>>>(width, height, imagef, image);

    //Copying the device memory image to device memory guesses
    for(int i = 0; i < numLayers; i++){
        hipMemcpy(&guess[i*width*height], image, width*height*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
        hipMemcpy(&u[i*width*height], image, width*height*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
    }

    for(int iter = 0; iter < iters; iter++){

        //Calculating the current iteration model
        for(int i = 0; i < numLayers; i++){
            placeHolder = &temporary[i*width*height];
            HplaceHolder = &Hq[i*width*height];
            propagate(HplaceHolder, image, placeHolder);
        }
        modelFunc<<<N_BLOCKS,N_THREADS>>>(width, height, numLayers, 1.0f, 0, temporary, model);
        ImodelFunc<<<N_BLOCKS,N_THREADS>>>(width,height,model,Imodel);

        placeHolder = &temporary[0];
        multiplyf<<<N_BLOCKS, N_THREADS>>>(width, height, Imodel, imagef, placeHolder);
        sum<<<N_BLOCKS, N_THREADS, N_THREADS*sizeof(float)>>>(width, height, placeHolder, sumArr);
        sum<<<1, N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS, 1, sumArr, sumArr);
        placeHolder = &temporary[count];
        multiplyf<<<N_BLOCKS, N_THREADS>>>(width, height, Imodel, Imodel, placeHolder);
        sum<<<N_BLOCKS, N_THREADS, N_THREADS*sizeof(float)>>>(width, height, placeHolder, &sumArr[N_BLOCKS]);
        sum<<<1, N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS, height, &sumArr[N_BLOCKS], &sumArr[N_BLOCKS]);

        simpleDivision<<<1,1>>>(imageSum, ImodelSum, c);

        linear<<<N_BLOCKS,N_THREADS>>>(width,height,c,imagef,Imodel, temporaryf);

        abs<<<N_BLOCKS,N_THREADS>>>(2*width,height,guess,&temporaryf[2*count]);
        sum<<<N_BLOCKS,N_THREADS, N_THREADS*sizeof(float)>>>(2*width,height,&temporaryf[2*count],sumArr);
        sum<<<1,N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS,sumArr,sumArr);

        square<<<N_BLOCKS,N_THREADS>>>(count, temporaryf, &temporaryf[count]);
        sum<<<N_BLOCKS,N_THREADS, N_THREADS*sizeof(float)>>>(2*width,height,&temporaryf[count],&sumArr[N_BLOCKS]);
        sum<<<1,N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS,&temporaryf[2*count],&sumArr[N_BLOCKS]);

        //Cost calculation with sparsity constraint
        cost<<<N_BLOCKS,N_THREADS>>>(width,height,numLayers,)
        
        
    
    }

    // Final cost calculation

    // Deallocation of variables
    hipFree(cost);
}

MultiLayer::~MultiLayer(){
    hipFree(Hq);
    hipFree(Hn);
    hipFree(temporary);
    hipFree(image);
    hipFree(imagef);
    hipFree(res);
    hipFree(model);
    hipFree(guess);
    hipFree(newGuess);
    hipFree(u);
    hipFree(Imodel);
    hipFree(sumArr);
    hipFree(temporaryf);
    hipFree(c);
    hipfftDestroy(fftPlan);
}
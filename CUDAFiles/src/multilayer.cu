#include "hip/hip_runtime.h"
#include "multilayer.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "kernels.h"
#include <vector>
#include <iostream>
#include <math.h>

MultiLayer::MultiLayer(int width, int height, std::vector<float> z, float dx, float lambda, float n) :width{width}, height{height}
{
    numLayers = (int)z.size();
    numBlocks = (width*height/2 + N_BLOCKS -1)/N_BLOCKS;

    hipMalloc(&Hq, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&Hn, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&res, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&guess, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&newGuess, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&u, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&temporary, numLayers*width*height*sizeof(hipfftComplex));
    hipMalloc(&temporaryf, 2*numLayers*width*height*sizeof(float));

    hipMalloc(&image, width*height*sizeof(hipfftComplex));
    hipMalloc(&imagef, width*height*sizeof(float));
    hipMalloc(&sumArr, 2*N_BLOCKS*sizeof(float));
    hipMalloc(&c, sizeof(float));
    
    hipMalloc(&model, width*height*sizeof(hipfftComplex));
    hipMalloc(&Imodel, width*height*sizeof(float));

    hipfftPlan2d(&fftPlan, width, height, HIPFFT_C2C);

    multilayerPropagator(z, dx, lambda, n);

}

void MultiLayer::multilayerPropagator(std::vector<float> z, float dx, float lambda, float n){
    hipfftComplex *placeHolder;
    for(int i = 0; i < numLayers; i++){
        placeHolder = &Hq[i*width*height];
        propagator<<<N_BLOCKS,N_THREADS>>>(width, height, z[i], dx, n, lambda, placeHolder);
    }
}

void MultiLayer::propagate(hipfftComplex* kernel, hipfftComplex* input, hipfftComplex* out){
    hipfftExecC2C(fftPlan, input, out, HIPFFT_FORWARD);
    multiply<<<N_BLOCKS, N_THREADS>>>(width,height,kernel,out);
    hipfftExecC2C(fftPlan, out, out, HIPFFT_BACKWARD);
}

void MultiLayer::iterate(float *input, int iters, float mu, float* rconstr, float* iconstr, float* modulus, float* phase){
    // Initialization of variables
    s = 1;
    float t = 0.5;
    int count = width*height;
    float *fplaceHolder;
    hipfftComplex *placeHolder;
    hipfftComplex *HplaceHolder;

    conjugate<<<N_BLOCKS,N_THREADS>>>(width*numLayers, height, Hq, Hn);

    //Allocating the device memory array for cost at each iteration
    hipMalloc(&cost, (1+iters)*sizeof(float));

    //Copying the input image from host to device memory - computationally complex
    hipMemcpy(imagef, input, width*height*sizeof(float), hipMemcpyHostToDevice);
    F2C<<<N_BLOCKS, N_THREADS>>>(width, height, imagef, image);

    //Copying the device memory image to device memory guesses
    for(int i = 0; i < numLayers; i++){
        hipMemcpy(&guess[i*width*height], image, width*height*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
        hipMemcpy(&u[i*width*height], image, width*height*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
    }

    for(int iter = 0; iter < iters; iter++){

        //Calculating the current iteration model
        for(int i = 0; i < numLayers; i++){
            placeHolder = &temporary[i*width*height];
            HplaceHolder = &Hq[i*width*height];
            propagate(HplaceHolder, image, placeHolder);
        }
        modelFunc<<<N_BLOCKS,N_THREADS>>>(width, height, numLayers, 1.0f, 0, temporary, model);
        ImodelFunc<<<N_BLOCKS,N_THREADS>>>(width,height,model,Imodel);

        fplaceHolder = temporaryf;
        multiplyf<<<N_BLOCKS, N_THREADS>>>(width, height, Imodel, imagef, fplaceHolder);
        sum<<<N_BLOCKS, N_THREADS, N_THREADS*sizeof(float)>>>(count, fplaceHolder, sumArr);
        sum<<<1, N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS, sumArr, sumArr);
        fplaceHolder = &temporaryf[count];
        multiplyf<<<N_BLOCKS, N_THREADS>>>(width, height, Imodel, Imodel, fplaceHolder);
        sum<<<N_BLOCKS, N_THREADS, N_THREADS*sizeof(float)>>>(count, fplaceHolder, &sumArr[N_BLOCKS]);
        sum<<<1, N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS, &sumArr[N_BLOCKS], &sumArr[N_BLOCKS]);

        simpleDivision<<<1,1>>>(sumArr, &sumArr[N_BLOCKS], c);

        linear<<<N_BLOCKS,N_THREADS>>>(width,height,c,imagef,Imodel, temporaryf, false);

        absolute<<<N_BLOCKS,N_THREADS>>>(numLayers*width,height,guess,&temporaryf[2*count]);
        sum<<<N_BLOCKS,N_THREADS, N_THREADS*sizeof(float)>>>(numLayers*count,&temporaryf[2*count],sumArr);
        sum<<<1,N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS,sumArr,sumArr);

        square<<<N_BLOCKS,N_THREADS>>>(count, temporaryf, &temporaryf[count]);
        sum<<<N_BLOCKS,N_THREADS, N_THREADS*sizeof(float)>>>(numLayers*count,&temporaryf[count],&sumArr[N_BLOCKS]);
        sum<<<1,N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS,&sumArr[N_BLOCKS],&sumArr[N_BLOCKS]);

        //Cost calculation with sparsity constraint
        cMultiplyf<<<1,1>>>(1,mu,sumArr,sumArr);
        simpleSum<<<1,1>>>(&sumArr[N_BLOCKS],sumArr,&cost[iter]);
        
        multiplyfc<<<N_BLOCKS,N_THREADS>>>(count*numLayers, temporaryf, temporary);
        for(int i = 0; i < numLayers; i++){
            placeHolder = &res[i*count];
            HplaceHolder = &Hn[i*count];
            propagate(HplaceHolder, temporary, placeHolder);
        }

        cMultiplyf<<<1,1>>>(1,(2*t),c,c);
        cMultiplyfcp<<<N_BLOCKS,N_THREADS>>>(numLayers*count, c, res, temporary);
        add<<<N_BLOCKS,N_THREADS>>>(count*numLayers, u, temporary, newGuess, false);

        //Applying strict bounds
        for(int i = 0 ; i < numLayers ; i++){
            placeHolder = &newGuess[count*i];
            strictBounds<<<N_BLOCKS,N_THREADS>>>(count, placeHolder, rconstr[i*2], rconstr[i*2+1], iconstr[i*2], iconstr[i*2+1]);
        }

        //Applying soft thresholding bounds
        softBounds<<<N_BLOCKS,N_THREADS>>>(count*numLayers,newGuess,mu,t);

        float s_new = 0.5*(1+std::sqrt(1+4*s*s));
        float temp = (s-1)/s_new;
        add<<<N_BLOCKS,N_THREADS>>>(count*numLayers,newGuess,guess, temporary,false);
        cMultiplyfc<<<N_BLOCKS,N_THREADS>>>(count,temp,temporary,temporary);
        add<<<N_BLOCKS,N_THREADS>>>(count*numLayers, newGuess, temporary, u, false);

        s = s_new;
        hipMemcpy(guess, newGuess, numLayers*width*height*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
    
    }

    // Final cost calculation
    //Calculating the current iteration model
    for(int i = 0; i < numLayers; i++){
        placeHolder = &temporary[i*width*height];
        HplaceHolder = &Hq[i*width*height];
        propagate(HplaceHolder, image, placeHolder);
    }
    modelFunc<<<N_BLOCKS,N_THREADS>>>(width, height, numLayers, 1.0f, 0, temporary, model);
    ImodelFunc<<<N_BLOCKS,N_THREADS>>>(width,height,model,Imodel);

    fplaceHolder = temporaryf;
    multiplyf<<<N_BLOCKS, N_THREADS>>>(width, height, Imodel, imagef, fplaceHolder);
    sum<<<N_BLOCKS, N_THREADS, N_THREADS*sizeof(float)>>>(count, fplaceHolder, sumArr);
    sum<<<1, N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS, sumArr, sumArr);
    fplaceHolder = &temporaryf[count];
    multiplyf<<<N_BLOCKS, N_THREADS>>>(width, height, Imodel, Imodel, fplaceHolder);
    sum<<<N_BLOCKS, N_THREADS, N_THREADS*sizeof(float)>>>(count, fplaceHolder, &sumArr[N_BLOCKS]);
    sum<<<1, N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS, &sumArr[N_BLOCKS], &sumArr[N_BLOCKS]);

    simpleDivision<<<1,1>>>(sumArr, &sumArr[N_BLOCKS], c);

    linear<<<N_BLOCKS,N_THREADS>>>(width,height,c,imagef,Imodel, temporaryf, false);

    absolute<<<N_BLOCKS,N_THREADS>>>(numLayers*width,height,guess,&temporaryf[2*count]);
    sum<<<N_BLOCKS,N_THREADS, N_THREADS*sizeof(float)>>>(numLayers*count,&temporaryf[2*count],sumArr);
    sum<<<1,N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS,sumArr,sumArr);

    square<<<N_BLOCKS,N_THREADS>>>(count, temporaryf, &temporaryf[count]);
    sum<<<N_BLOCKS,N_THREADS, N_THREADS*sizeof(float)>>>(numLayers*count,&temporaryf[count],&sumArr[N_BLOCKS]);
    sum<<<1,N_BLOCKS, N_BLOCKS*sizeof(float)>>>(N_BLOCKS,&sumArr[N_BLOCKS],&sumArr[N_BLOCKS]);

    //Cost calculation with sparsity constraint
    cMultiplyf<<<1,1>>>(1,mu,sumArr,sumArr);
    simpleSum<<<1,1>>>(&sumArr[N_BLOCKS],sumArr,&cost[iters]);

    // Moving results to host memory

    absolute<<<N_BLOCKS,N_THREADS>>>(width*numLayers,height,guess,temporaryf);
    hipMemcpy(modulus,temporaryf,width*numLayers*height,hipMemcpyDeviceToHost);
    angle<<<N_BLOCKS,N_THREADS>>>(count*numLayers,guess,temporaryf);
    hipMemcpy(phase,temporaryf,width*numLayers*height,hipMemcpyDeviceToHost);

    // Deallocation of variables
    hipFree(cost);
}

MultiLayer::~MultiLayer(){
    hipFree(Hq);
    hipFree(Hn);
    hipFree(temporary);
    hipFree(image);
    hipFree(imagef);
    hipFree(res);
    hipFree(model);
    hipFree(guess);
    hipFree(newGuess);
    hipFree(u);
    hipFree(temporaryf);
    hipFree(c);
    hipfftDestroy(fftPlan);
    free(h_cost);
}
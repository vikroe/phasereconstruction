#include "hip/hip_runtime.h"
#include "multilayer.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "kernels.h"
#include <vector>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include "stdio.h"
#include "cudaDebug.h"
#include "blur.h"

MultiLayer::MultiLayer(int width, 
                        int height, 
                        std::vector<double> z,
                        std::vector<double> rconstr, 
                        std::vector<double> iconstr, 
                        double mu, 
                        double dx, 
                        double lambda, 
                        double n) :width(width), height(height), z(z), rconstr(rconstr), iconstr(iconstr), mu(mu)
{
    numLayers = (int)z.size();
    count = width*height;
    m_count = width*height*numLayers;
    blur = new Blur();

    allocate();
    multilayerPropagator(dx, lambda, n);
    conjugate<<<N_BLOCKS,N_THREADS>>>(m_count, Hq, Hn);
}

void MultiLayer::allocate(){
    hipMalloc(&model, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&Hq, m_count*sizeof(hipfftComplex));
    hipMalloc(&Hn, m_count*sizeof(hipfftComplex));
    hipMalloc(&propagation, m_count*sizeof(hipfftComplex));
    hipMalloc(&guess, m_count*sizeof(hipfftDoubleComplex));
    hipMalloc(&newGuess, m_count*sizeof(hipfftDoubleComplex));
    hipMalloc(&u, m_count*sizeof(hipfftDoubleComplex));
    hipMalloc(&temporary, m_count*sizeof(hipfftDoubleComplex));
    hipMalloc(&sumArr, 2*N_BLOCKS*sizeof(double));
    hipMalloc(&c, sizeof(double));
    hipMalloc(&image, count*sizeof(double));
    hipMalloc(&Imodel, count*sizeof(double));
    hipMalloc(&temporaryf, 2*m_count*sizeof(double));
    
    int n[2] = {height, width};
    hipfftPlanMany(&fftPlan, 2, n, NULL, 1, count, NULL, 1, count, HIPFFT_C2C, 2);
}

void MultiLayer::multilayerPropagator(double dx, double lambda, double n){
    hipfftComplex *placeHolder;
    for(int i = 0; i < numLayers; i++){
        placeHolder = &Hq[i*count];
        propagator<<<N_BLOCKS,N_THREADS>>>(width, height, z[i], dx, n, lambda, placeHolder);
    }
}

void MultiLayer::propagate(hipfftComplex* kernel, hipfftDoubleComplex* input, hipfftDoubleComplex* out){
    Z2C<<<N_BLOCKS,N_THREADS>>>(m_count, input, propagation);
    hipfftExecC2C(fftPlan, propagation, propagation, HIPFFT_FORWARD);
    multiply<<<N_BLOCKS, N_THREADS>>>(m_count, kernel, propagation);
    hipfftExecC2C(fftPlan, propagation, propagation, HIPFFT_BACKWARD);
    C2Z<<<N_BLOCKS,N_THREADS>>>(m_count, propagation, out);
}

void MultiLayer::calculateCost(double mu, double* model, hipfftDoubleComplex* guess, double* temp, double* out){
    absolute<<<N_BLOCKS,N_THREADS>>>(m_count, guess, &temp[m_count]);
    square<<<N_BLOCKS,N_THREADS>>>(count, model, &temp[count]);

    h_sum(m_count, &temp[m_count], sumArr);
    h_sum(count, &temp[count], &sumArr[N_BLOCKS]);
    
    scalef<<<1,1>>>(1,mu,sumArr,sumArr);
    simpleSum<<<1,1>>>(&sumArr[N_BLOCKS],sumArr,&out[0]);
}

void MultiLayer::normalize(int cnt, double* arr){
    h_minimum(cnt, arr, sumArr);

    double temp;
    hipMemcpy(&temp, sumArr, sizeof(double), hipMemcpyDeviceToHost);
    offsetf<<<N_BLOCKS,N_THREADS>>>(cnt, -temp, arr, arr, true);

    h_maximum(cnt, arr, sumArr);
    contractf_p<<<N_BLOCKS,N_THREADS>>>(cnt, sumArr, arr, arr);
}

void MultiLayer::iterate(double *input, int iters, bool b_cost, bool warm){
    // Initialization of variables
    s = 1;
    if(b_cost){
        hipMalloc(&cost, (1+iters)*sizeof(double));
        h_cost = (double*)malloc((iters+1)*sizeof(double));
    }

    //Copying the input image from host to device memory - computationally complex
    gpuErrchk(hipMemcpy(image, input, count*sizeof(double), hipMemcpyHostToDevice));
    blur->gaussianBlur(width,height, 5, 3, image, temporaryf, image);

    //Copying the device memory image to device memory guesses

    for(int i = 0; i < numLayers; i++){
        F2C<<<N_BLOCKS,N_THREADS>>>(count, image, &u[i*count]);
        if (!warm)
            F2C<<<N_BLOCKS,N_THREADS>>>(count, image, &guess[i*count]);
    }

    for(int iter = 0; iter < iters; iter++){
        //Calculating the current iteration model 
        propagate(Hq, u, temporary);

        //Calculation of Imodel and model arrays
        modelFunc<<<N_BLOCKS,N_THREADS>>>(count, numLayers, 1.0f, 0, temporary, model, Imodel);

        //Calculation of the optimal scaling parameter c
        h_sumOfProducts(count, image, Imodel, sumArr);
        h_sumOfProducts(count, Imodel, Imodel, &sumArr[N_BLOCKS]);
        contractf_p<<<1,1>>>(1, &sumArr[N_BLOCKS], sumArr, c);
        double t_cost[1];
        hipMemcpy(t_cost, c, sizeof(double), hipMemcpyDeviceToHost);
        std::cout << "Current optimal scaling factor is " << t_cost[0] << std::endl;

        //Cost calculation with sparsity constraint
        linear<<<N_BLOCKS,N_THREADS>>>(count, c, image, Imodel, temporaryf, false);

        if(b_cost){
            calculateCost(mu, temporaryf, guess, temporaryf, &cost[iter]);
            double t_cost[1];
            hipMemcpy(t_cost, &cost[iter], sizeof(double), hipMemcpyDeviceToHost);
            std::cout << t_cost[0] << std::endl;
        }

        //Calculating residues
        multiplyfc<<<N_BLOCKS,N_THREADS>>>(count, temporaryf, model);
        extend<<<N_BLOCKS,N_THREADS>>>(count, numLayers, model, temporary);
        propagate(Hn, temporary, temporary);

        F2C<<<1,1>>>(1,c,newGuess);
        scale_p<<<N_BLOCKS,N_THREADS>>>(m_count, newGuess, temporary, temporary);
        add<<<N_BLOCKS,N_THREADS>>>(m_count, u, temporary, newGuess, false);

        //Applying strict bounds
        for(int i = 0 ; i < numLayers ; i++){
            strictBounds<<<N_BLOCKS,N_THREADS>>>(count, &newGuess[count*i], rconstr[i*2], rconstr[i*2+1], iconstr[i*2], iconstr[i*2+1]);
        }

        //Applying soft thresholding bounds
        softBounds<<<N_BLOCKS,N_THREADS>>>(m_count, newGuess, mu, 0.5f);

        double s_new = 0.5*(1+std::sqrt(1+4*s*s));
        hipfftDoubleComplex temp = make_hipDoubleComplex((s-1)/s_new,0);
        add<<<N_BLOCKS,N_THREADS>>>(m_count, newGuess, guess, temporary, false);
        scale<<<N_BLOCKS,N_THREADS>>>(m_count, temp, temporary, temporary);
        add<<<N_BLOCKS,N_THREADS>>>(m_count, newGuess, temporary, u, true);

        s = s_new;
        hipMemcpy(guess, newGuess, m_count*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToDevice);
    
    }
    
    // Final cost calculation
    if(b_cost){
        propagate(Hq, u, newGuess);

        //Calculation of Imodel and model arrays
        modelFunc<<<N_BLOCKS,N_THREADS>>>(count, numLayers, 1.0f, 0, newGuess, model, Imodel);

        //Calculation of the optimal scaling parameter c
        h_sumOfProducts(count, image, Imodel, sumArr);
        h_sumOfProducts(count, Imodel, Imodel, &sumArr[N_BLOCKS]);
        contractf_p<<<1,1>>>(1, &sumArr[N_BLOCKS], sumArr, c);

        //Cost calculation with sparsity constraint
        linear<<<N_BLOCKS,N_THREADS>>>(count, c, image, Imodel, temporaryf, false);

        calculateCost(mu, temporaryf, guess, temporaryf, &cost[iters]);
        double t_cost[1];
        hipMemcpy(t_cost, &cost[iters], sizeof(double), hipMemcpyDeviceToHost);
        std::cout << t_cost[0] << std::endl;

        gpuErrchk(hipMemcpy(h_cost, cost, (iters+1)*sizeof(double), hipMemcpyDeviceToHost));
        hipFree(cost);
    }

    // Moving results to host memory
    // Adding one to get the light wavefront (otherwise we only have the disturbance by the particles and electrodes)
    offset<<<N_BLOCKS,N_THREADS>>>(m_count, 1.0f, 0.0f, guess, temporary);

    // Check if any error occured - important to note that untested kernels can lead to exceptions at hipMemcpy calls
    gpuErrchk(hipPeekAtLastError());
}

void MultiLayer::update(uint8_t* modulus, uint8_t* phase){
    // temporary contains the latest results in complex form
    
    // Processing the modulus of both layers
    absolute<<<N_BLOCKS,N_THREADS>>>(m_count,temporary,temporaryf);
    for(int i = 0; i < numLayers; i++){
        normalize(count, &temporaryf[i*count]);
    }
    D2u8<<<N_BLOCKS,N_THREADS>>>(m_count,temporaryf,modulus);

    // Processing the phase of both layers
    angle<<<N_BLOCKS,N_THREADS>>>(m_count,temporary,temporaryf);
    for(int i = 0; i < numLayers; i++){
        normalize(count, &temporaryf[i*count]);
    }
    D2u8<<<N_BLOCKS,N_THREADS>>>(m_count,temporaryf,phase);
}

MultiLayer::~MultiLayer(){
    hipFree(Hq);
    hipFree(Hn);
    hipFree(temporary);
    hipFree(image);
    hipFree(model);
    hipFree(guess);
    hipFree(newGuess);
    hipFree(u);
    hipFree(temporaryf);
    hipFree(c);
    hipFree(propagation);
    hipfftDestroy(fftPlan);
}
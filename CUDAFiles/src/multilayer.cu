#include "hip/hip_runtime.h"
#include "multilayer.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "kernels.h"
#include <vector>
#include <iostream>
#include <math.h>
#include <cuda_device_runtime_api.h>
#include <assert.h>
#include "stdio.h"
#include "string.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stdout,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void cudaMemoryTest()
{
    const unsigned int N = 1048576;
    const unsigned int bytes = N * sizeof(int);
    int *h_a = (int*)malloc(bytes);
    int *d_a;
    gpuErrchk(hipMalloc((int**)&d_a, bytes));

    memset(h_a, 0, bytes);
    gpuErrchk(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));
}

MultiLayer::MultiLayer(int width, int height, std::vector<float> z, float dx, float lambda, float n) :width{width}, height{height}
{
    numLayers = (int)z.size();
    count = width*height;
    m_count = width*height*numLayers;

    hipMalloc(&Hq, m_count*sizeof(hipfftComplex));
    hipMalloc(&Hn, m_count*sizeof(hipfftComplex));
    hipMalloc(&res, m_count*sizeof(hipfftComplex));
    hipMalloc(&guess, m_count*sizeof(hipfftComplex));
    hipMalloc(&newGuess, m_count*sizeof(hipfftComplex));
    hipMalloc(&u, m_count*sizeof(hipfftComplex));
    hipMalloc(&temporary, m_count*sizeof(hipfftComplex));
    hipMalloc(&temporaryf, 2*m_count*sizeof(float));

    hipMalloc(&image, count*sizeof(hipfftComplex));
    hipMalloc(&imagef, count*sizeof(float));
    hipMalloc(&sumArr, 2*N_BLOCKS*sizeof(float));
    hipMalloc(&c, sizeof(float));
    
    hipMalloc(&model, count*sizeof(hipfftComplex));
    hipMalloc(&Imodel, count*sizeof(float));

    modulus = (float*)malloc(m_count*sizeof(float));
    phase = (float*)malloc(m_count*sizeof(float));

    hipfftPlan2d(&fftPlan, width, height, HIPFFT_C2C);

    multilayerPropagator(z, dx, lambda, n);

}

void MultiLayer::multilayerPropagator(std::vector<float> z, float dx, float lambda, float n){
    hipfftComplex *placeHolder;
    for(int i = 0; i < numLayers; i++){
        placeHolder = &Hq[i*count];
        propagator<<<N_BLOCKS,N_THREADS>>>(width, height, z[i], dx, n, lambda, placeHolder);
        gpuErrchk(hipPeekAtLastError());
    }
}

void MultiLayer::propagate(hipfftComplex* kernel, hipfftComplex* input, hipfftComplex* out){
    hipfftExecC2C(fftPlan, input, out, HIPFFT_FORWARD);
    multiply<<<N_BLOCKS, N_THREADS>>>(count,kernel,out);
    hipfftExecC2C(fftPlan, out, out, HIPFFT_BACKWARD);
    gpuErrchk(hipPeekAtLastError());
}

void MultiLayer::iterate(float *input, int iters, float mu, float* rconstr, float* iconstr){
    // Initialization of variables
    s = 1;
    float t = 0.5;
    float *fplaceHolder;
    h_cost = (float*)malloc((iters+1)*sizeof(float));
    hipfftComplex *placeHolder;
    hipfftComplex *HplaceHolder;

    conjugate<<<N_BLOCKS,N_THREADS>>>(m_count, Hq, Hn);

    //Allocating the device memory array for cost at each iteration
    hipMalloc(&cost, (1+iters)*sizeof(float));

    //Copying the input image from host to device memory - computationally complex
    hipMemcpy(imagef, input, count*sizeof(float), hipMemcpyHostToDevice);
    F2C<<<N_BLOCKS, N_THREADS>>>(count, imagef, image);

    //Copying the device memory image to device memory guesses
    for(int i = 0; i < numLayers; i++){
        hipMemcpy(&guess[i*count], image, count*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
        hipMemcpy(&u[i*count], image, count*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
    }
    gpuErrchk(hipPeekAtLastError());

    for(int iter = 0; iter < iters; iter++){

        //Calculating the current iteration model
        for(int i = 0; i < numLayers; i++){
            placeHolder = &temporary[i*count];
            HplaceHolder = &Hq[i*count];
            propagate(HplaceHolder, image, placeHolder);
        }
        modelFunc<<<N_BLOCKS,N_THREADS>>>(count, numLayers, 1.0f, 0, temporary, model);
        ImodelFunc<<<N_BLOCKS,N_THREADS>>>(count, model, Imodel);

        fplaceHolder = temporaryf;
        multiplyf<<<N_BLOCKS, N_THREADS>>>(count, Imodel, imagef, fplaceHolder);
        sum<<<1, N_THREADS, N_THREADS*sizeof(float)>>>(count, fplaceHolder, sumArr);

        fplaceHolder = &temporaryf[count];
        multiplyf<<<N_BLOCKS, N_THREADS>>>(count, Imodel, Imodel, fplaceHolder);
        sum<<<1, N_THREADS, N_THREADS*sizeof(float)>>>(count, fplaceHolder, &sumArr[N_BLOCKS]);

        simpleDivision<<<1,1>>>(sumArr, &sumArr[N_BLOCKS], c);
        cudaMemoryTest();

        linear<<<N_BLOCKS,N_THREADS>>>(count, c, imagef, Imodel, temporaryf, false);

        absolute<<<N_BLOCKS,N_THREADS>>>(m_count,guess,&temporaryf[2*count]);
        sum<<<1,N_THREADS, N_THREADS*sizeof(float)>>>(m_count,&temporaryf[2*count],sumArr);

        square<<<N_BLOCKS,N_THREADS>>>(count, temporaryf, &temporaryf[count]);
        sum<<<1,N_THREADS, N_THREADS*sizeof(float)>>>(m_count,&temporaryf[count],&sumArr[N_BLOCKS]);

        //Cost calculation with sparsity constraint
        cMultiplyf<<<1,1>>>(1,mu,sumArr,sumArr);
        simpleSum<<<1,1>>>(&sumArr[N_BLOCKS],sumArr,&cost[iter]);
        
        multiplyfc<<<N_BLOCKS,N_THREADS>>>(m_count, temporaryf, temporary);
        for(int i = 0; i < numLayers; i++){
            placeHolder = &res[i*count];
            HplaceHolder = &Hn[i*count];
            propagate(HplaceHolder, temporary, placeHolder);
        }

        cMultiplyf<<<1,1>>>(1,(2*t),c,c);
        cMultiplyfcp<<<N_BLOCKS,N_THREADS>>>(m_count, c, res, temporary);
        add<<<N_BLOCKS,N_THREADS>>>(m_count, u, temporary, newGuess, false);

        //Applying strict bounds
        for(int i = 0 ; i < numLayers ; i++){
            placeHolder = &newGuess[count*i];
            strictBounds<<<N_BLOCKS,N_THREADS>>>(count, placeHolder, rconstr[i*2], rconstr[i*2+1], iconstr[i*2], iconstr[i*2+1]);
        }

        //Applying soft thresholding bounds
        softBounds<<<N_BLOCKS,N_THREADS>>>(m_count, newGuess, mu, t);

        float s_new = 0.5*(1+std::sqrt(1+4*s*s));
        float temp = (s-1)/s_new;
        add<<<N_BLOCKS,N_THREADS>>>(m_count, newGuess, guess, temporary, false);
        cMultiplyfc<<<N_BLOCKS,N_THREADS>>>(count, temp, temporary, temporary);
        add<<<N_BLOCKS,N_THREADS>>>(m_count, newGuess, temporary, u, false);

        s = s_new;
        hipMemcpy(guess, newGuess, m_count*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
    
    }

    // Final cost calculation
    //Calculating the current iteration model
    for(int i = 0; i < numLayers; i++){
        placeHolder = &temporary[i*width*height];
        HplaceHolder = &Hq[i*width*height];
        propagate(HplaceHolder, image, placeHolder);
    }
    modelFunc<<<N_BLOCKS,N_THREADS>>>(count, numLayers, 1.0f, 0, temporary, model);
    ImodelFunc<<<N_BLOCKS,N_THREADS>>>(count, model, Imodel);

    fplaceHolder = temporaryf;
    multiplyf<<<N_BLOCKS, N_THREADS>>>(count, Imodel, imagef, fplaceHolder);
    sum<<<1, N_THREADS, N_THREADS*sizeof(float)>>>(count, fplaceHolder, sumArr);
    fplaceHolder = &temporaryf[count];
    multiplyf<<<N_BLOCKS, N_THREADS>>>(count, Imodel, Imodel, fplaceHolder);
    sum<<<1, N_THREADS, N_THREADS*sizeof(float)>>>(count, fplaceHolder, &sumArr[N_BLOCKS]);
    gpuErrchk(hipPeekAtLastError());

    simpleDivision<<<1,1>>>(sumArr, &sumArr[N_BLOCKS], c);

    linear<<<N_BLOCKS,N_THREADS>>>(count, c, imagef, Imodel, temporaryf, false);

    absolute<<<N_BLOCKS,N_THREADS>>>(m_count,guess,&temporaryf[2*count]);
    sum<<<1,N_THREADS, N_THREADS*sizeof(float)>>>(m_count,&temporaryf[2*count],sumArr);

    square<<<N_BLOCKS,N_THREADS>>>(count, temporaryf, &temporaryf[count]);
    sum<<<1,N_THREADS, N_THREADS*sizeof(float)>>>(m_count,&temporaryf[count],&sumArr[N_BLOCKS]);

    //Cost calculation with sparsity constraint
    cMultiplyf<<<1,1>>>(1,mu,sumArr,sumArr);
    simpleSum<<<1,1>>>(&sumArr[N_BLOCKS],sumArr,&cost[iters]);
    gpuErrchk(hipPeekAtLastError());

    // Moving results to host memory

    absolute<<<N_BLOCKS,N_THREADS>>>(m_count,guess,temporaryf);
    maximum<<<1,N_THREADS,N_THREADS*sizeof(float)>>>(m_count, temporaryf, sumArr);
    cDividefp<<<N_BLOCKS,N_THREADS>>>(m_count,sumArr,temporaryf,temporaryf);
    gpuErrchk(hipMemcpy(modulus,temporaryf,m_count*sizeof(float),hipMemcpyDeviceToHost));
    angle<<<N_BLOCKS,N_THREADS>>>(m_count,guess,temporaryf);
    gpuErrchk(hipMemcpy(phase,temporaryf,m_count*sizeof(float),hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_cost, cost, iters+1, hipMemcpyDeviceToHost));

    printf("Did we get here at all?\n");

    // Deallocation of variables
    hipFree(cost);
    gpuErrchk(hipPeekAtLastError());

}

MultiLayer::~MultiLayer(){
    hipFree(Hq);
    hipFree(Hn);
    hipFree(temporary);
    hipFree(image);
    hipFree(imagef);
    hipFree(res);
    hipFree(model);
    hipFree(guess);
    hipFree(newGuess);
    hipFree(u);
    hipFree(temporaryf);
    hipFree(c);
    hipfftDestroy(fftPlan);
    free(h_cost);
    free(modulus);
    free(phase);
}
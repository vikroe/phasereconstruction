#include "hip/hip_runtime.h"
#include "fista.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "kernels.h"
#include <vector>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include "stdio.h"
#include "cudaDebug.h"
#include "blur.h"

Fista::Fista(
            double z,
            std::vector<double> rconstr,
            std::vector<double> iconstr,
            double mu,
            int width,
            int height,
            bool b_cost,
            double dx,
            double lambda,
            double n
): width(width), height(height), b_cost(b_cost), mu(mu), rconstr(rconstr), iconstr(iconstr), z(z)
{

    count = width*height;
    blur = new Blur();

    allocate();
    propagator<<<N_BLOCKS, N_THREADS>>>(width, height, z, dx, n, lambda, Hq);
    conjugate<<<N_BLOCKS,N_THREADS>>>(count, Hq, Hn);
}

void Fista::allocate(){
    hipMalloc(&model, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&Hq, count*sizeof(hipfftComplex));
    hipMalloc(&Hn, count*sizeof(hipfftComplex));
    hipMalloc(&propagation, count*sizeof(hipfftComplex));
    hipMalloc(&guess, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&newGuess, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&u, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&temporary, count*sizeof(hipfftDoubleComplex));
    hipMalloc(&sumArr, 2*N_BLOCKS*sizeof(double));
    hipMalloc(&c, sizeof(double));
    hipMalloc(&image, count*sizeof(double));
    hipMalloc(&Imodel, count*sizeof(double));
    hipMalloc(&temporaryf, 2*count*sizeof(double));
    hipfftPlan2d(&fftPlan, height, width, HIPFFT_C2C);
}

void Fista::propagate(hipfftComplex* kernel, hipfftDoubleComplex* input, hipfftDoubleComplex* out){
    Z2C<<<N_BLOCKS,N_THREADS>>>(count, input, propagation);
    hipfftExecC2C(fftPlan, propagation, propagation, HIPFFT_FORWARD);
    multiply<<<N_BLOCKS, N_THREADS>>>(count, kernel, propagation);
    hipfftExecC2C(fftPlan, propagation, propagation, HIPFFT_BACKWARD);
    C2Z<<<N_BLOCKS,N_THREADS>>>(count, propagation, out);
}

void Fista::calculateCost(double mu, double* model, hipfftDoubleComplex* guess, double* temp, double* out){
    absolute<<<N_BLOCKS,N_THREADS>>>(count, guess, &temp[count]);
    square<<<N_BLOCKS,N_THREADS>>>(count, model, &temp[count]);

    h_sum(count, &temp[count], sumArr);
    h_sum(count, &temp[count], &sumArr[N_BLOCKS]);
    
    scalef<<<1,1>>>(1,mu,sumArr,sumArr);
    simpleSum<<<1,1>>>(&sumArr[N_BLOCKS],sumArr,&out[0]);
}

void Fista::normalize(int c, double* arr){
    h_minimum(c, arr, sumArr);

    double temp;
    hipMemcpy(&temp, sumArr, sizeof(double), hipMemcpyDeviceToHost);
    offsetf<<<N_BLOCKS,N_THREADS>>>(c, -temp, arr, arr, true);

    h_maximum(c, arr, sumArr);
    contractf_p<<<N_BLOCKS,N_THREADS>>>(c, sumArr, arr, arr);
}

void Fista::iterate(double *input, int iters, bool warm){
    // Initialization of variables
    s = 1;
    if(b_cost){
        hipMalloc(&cost, (1+iters)*sizeof(double));
        h_cost = (double*)malloc((iters+1)*sizeof(double));
    }

    //Copying the input image from host to device memory - computationally complex
    gpuErrchk(hipMemcpy(image, input, count*sizeof(double), hipMemcpyHostToDevice));
    blur->gaussianBlur(width,height, 5, 3, image, temporaryf, image);
    h_average(count, image, sumArr);
    contractf_p<<<N_BLOCKS,N_THREADS>>>(count, sumArr, image, image);
    hipMemcpy(m, sumArr, sizeof(double), hipMemcpyDeviceToHost);

    //Copying the device memory image to device memory guesses

    F2C<<<N_BLOCKS,N_THREADS>>>(count, image, u);
    if (!warm)
        F2C<<<N_BLOCKS,N_THREADS>>>(count, image, guess);
    
    for(int iter = 0; iter < iters; iter++){
        //Calculating the current iteration model 
        propagate(Hq, u, temporary);

        //Calculation of Imodel and model arrays
        modelFunc<<<N_BLOCKS,N_THREADS>>>(count, 1.0f, 0, temporary, model, Imodel);

        //Calculation of the optimal scaling parameter c
        h_sumOfProducts(count, image, Imodel, sumArr);
        h_sumOfProducts(count, Imodel, Imodel, &sumArr[N_BLOCKS]);
        contractf_p<<<1,1>>>(1, &sumArr[N_BLOCKS], sumArr, c);
        double t_cost[1];
        hipMemcpy(t_cost, c, sizeof(double), hipMemcpyDeviceToHost);

        //Cost calculation with sparsity constraint
        linear<<<N_BLOCKS,N_THREADS>>>(count, c, image, Imodel, temporaryf, false);

        if(b_cost){
            calculateCost(mu, temporaryf, guess, temporaryf, &cost[iter]);
            double t_cost[1];
            hipMemcpy(t_cost, &cost[iter], sizeof(double), hipMemcpyDeviceToHost);
            std::cout << "[DEBUG] Cost at iteration " << iter << " is " << t_cost[0] << std::endl;
        }

        //Calculating residues
        multiplyfc<<<N_BLOCKS,N_THREADS>>>(count, temporaryf, model);
        propagate(Hn, model, temporary);

        double t = 0.22;
        scalef<<<1,1>>>(1, 2*t, c, c);
        F2C<<<1,1>>>(1,c,newGuess);
        scale_p<<<N_BLOCKS,N_THREADS>>>(count, newGuess, temporary, temporary);
        add<<<N_BLOCKS,N_THREADS>>>(count, u, temporary, newGuess, false);

        //Applying soft thresholding bounds
        softBounds<<<N_BLOCKS,N_THREADS>>>(count, newGuess, mu, t);

        //Applying strict bounds
        strictBounds<<<N_BLOCKS,N_THREADS>>>(count, newGuess, rconstr[0], rconstr[1], iconstr[0], iconstr[1]);

        double s_new = 0.5*(1+std::sqrt(1+4*s*s));
        hipfftDoubleComplex temp = make_hipDoubleComplex((s-1)/s_new,0);
        add<<<N_BLOCKS,N_THREADS>>>(count, newGuess, guess, temporary, false);
        scale<<<N_BLOCKS,N_THREADS>>>(count, temp, temporary, temporary);
        add<<<N_BLOCKS,N_THREADS>>>(count, newGuess, temporary, u, true);

        s = s_new;
        hipMemcpy(guess, newGuess, count*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToDevice);
    
    }
    
    // Final cost calculation
    if(b_cost){
        propagate(Hq, u, newGuess);

        //Calculation of Imodel and model arrays
        modelFunc<<<N_BLOCKS,N_THREADS>>>(count, 1.0f, 0, newGuess, model, Imodel);

        //Calculation of the optimal scaling parameter c
        h_sumOfProducts(count, image, Imodel, sumArr);
        h_sumOfProducts(count, Imodel, Imodel, &sumArr[N_BLOCKS]);
        contractf_p<<<1,1>>>(1, &sumArr[N_BLOCKS], sumArr, c);

        //Cost calculation with sparsity constraint
        linear<<<N_BLOCKS,N_THREADS>>>(count, c, image, Imodel, temporaryf, false);

        calculateCost(mu, temporaryf, guess, temporaryf, &cost[iters]);
        double t_cost[1];
        hipMemcpy(t_cost, &cost[iters], sizeof(double), hipMemcpyDeviceToHost);
        std::cout << "Current cost at iteration " << iters << " is " << t_cost[0] << std::endl;

        gpuErrchk(hipMemcpy(h_cost, cost, (iters+1)*sizeof(double), hipMemcpyDeviceToHost));
        hipFree(cost);
    }

    // Moving results to host memory
    // Adding one to get the light wavefront (otherwise we only have the disturbance by the particles and electrodes)
    offset<<<N_BLOCKS,N_THREADS>>>(count, 1.0f, 0.0f, guess, temporary);

    // Check if any error occured - important to note that untested kernels can lead to exceptions at hipMemcpy calls
    gpuErrchk(hipPeekAtLastError());
}

void Fista::update(uint8_t* modulus, uint8_t* phase){
    // temporary contains the latest results in complex form
    
    // Processing the modulus
    absolute<<<N_BLOCKS,N_THREADS>>>(count,temporary,temporaryf);
    scalef<<<N_BLOCKS,N_THREADS>>>(count, m[0], temporaryf, temporaryf);
    //normalize(count, temporaryf);
    D2u8<<<N_BLOCKS,N_THREADS>>>(count,temporaryf,modulus);

    // Processing the phase
    angle<<<N_BLOCKS,N_THREADS>>>(count,temporary,temporaryf);
    normalize(count, temporaryf);
    D2u8<<<N_BLOCKS,N_THREADS>>>(count,temporaryf,phase);
}

Fista::~Fista(){
    hipFree(Hq);
    hipFree(Hn);
    hipFree(temporary);
    hipFree(image);
    hipFree(model);
    hipFree(guess);
    hipFree(newGuess);
    hipFree(u);
    hipFree(temporaryf);
    hipFree(c);
    hipFree(propagation);
    hipfftDestroy(fftPlan);
}